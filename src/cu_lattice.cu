
#include <hip/hip_runtime.h>
///*
// * cu_lattice.cu
// *
// *  Created on: Apr 4, 2016
// *      Author: Kerstin Vater
// * Description: This class defines a lattice gas cellular automaton in two
// *              dimensions parallelized by means of Nvidia CUDA.
// */

//#include "cu_lattice.h"

//__device__ Real cu_random(int seed, int offset) {

//    // CUDA's random number library uses curandState_t to keep
//    // track of the seed value we will store a random state for
//    // every thread.
//    curandState_t state;

//    // We have to initialize the state.
//    curand_init(seed,    // The seed controls the sequence of random values that are produced.
//                0,       // The sequence number is only important with multiple cores.
//                offset,  // The offset is how much extra we advance in the sequence for each call, can be 0.
//                &state);

//    return curand_uniform(&state);
//}

//__device__ bool cu_random_bool(int seed, int offset) {

//    return (cu_random(seed, offset) > 0.5);
//}

//// CUDA kernel which performs the collision and propagation step
//// on the lattice gas automaton.
//template <int n_dir>
//__global__ void collide_and_propagate_kernel(const int   n_x,
//                                             const int   n_y,
//                                             const int   n_cells,
//                                                   char* cell_type_gpu,
//                                                   char* node_state_gpu,
//                                                   char* node_state_tmp_gpu,
//                                             unsigned int step) {

//#ifdef DEBUG
//            // Check weather the domain dimensions are valid for the FHP model.
//            if (n_y % 2 != 0 && n_dir == 6) {

//                printf("ERROR in collide_and_propagate_kernel(): "
//                       "Invalid domain dimension in y direction.\n");
//                abort();
//            }
//#endif

//    // Each thread is working on one cell of the lattice.

//    // Thread is active per default.
//    bool active = true;

//    // Calculate the position of the cell in x direction (column index).
//    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;

//    // Calculate the position of the cell in y direction (row index).
//    int pos_y = blockIdx.y;

//    // Check weather the thread is working on a valid cell.
//    if (pos_x >= n_x) {

//        active = false;
//    }

//    // Start calculation only for activated threads working on valid cells.
//    if (active) {

//        // Memory offset to neighbor cells in the different directions for the
//        // propagation step.
//        // Note that for the FHP model there is a difference in the offsets depending
//        // on weather the cell is located in a row with even or odd index.
//        int offset_to_neighbor[n_dir];

//        // Memory offset to related cells of the opposite boundary in the different
//        // directions in case of periodic boundaries.
//        int offset_to_eastern_boundary[n_dir];
//        int offset_to_northern_boundary[n_dir];
//        int offset_to_western_boundary[n_dir];
//        int offset_to_southern_boundary[n_dir];

//        // Inverse direction indices for each lattice direction.
//        char inverse_dir[n_dir];

//        // Mirrored direction indices for each lattice direction with respect
//        // to the x and y axis.
//        char mirrored_dir_x[n_dir];
//        char mirrored_dir_y[n_dir];

//        // Set the components of the lattice vectors for the different directions.
//        //
//        // Set the model based values according to the number of lattice directions.
//        switch (n_dir) {

//            // HPP model.
//            case 4:
//            {
//                offset_to_neighbor[0] = 1;
//                offset_to_neighbor[1] = n_x;
//                offset_to_neighbor[2] = -1;
//                offset_to_neighbor[3] = -n_x;

//                offset_to_eastern_boundary[0] = 0;
//                offset_to_eastern_boundary[1] = 0;
//                offset_to_eastern_boundary[2] = n_x;
//                offset_to_eastern_boundary[3] = 0;

//                offset_to_northern_boundary[0] = 0;
//                offset_to_northern_boundary[1] = 0;
//                offset_to_northern_boundary[2] = 0;
//                offset_to_northern_boundary[3] = n_x * n_y;

//                offset_to_western_boundary[0] = -n_x;
//                offset_to_western_boundary[1] = 0;
//                offset_to_western_boundary[2] = 0;
//                offset_to_western_boundary[3] = 0;

//                offset_to_southern_boundary[0] = 0;
//                offset_to_southern_boundary[1] = -n_x * n_y;
//                offset_to_southern_boundary[2] = 0;
//                offset_to_southern_boundary[3] = 0;

//                inverse_dir[0] = 2;
//                inverse_dir[1] = 3;
//                inverse_dir[2] = 0;
//                inverse_dir[3] = 1;

//                mirrored_dir_x[0] = 0;
//                mirrored_dir_x[1] = 3;
//                mirrored_dir_x[2] = 2;
//                mirrored_dir_x[3] = 1;

//                mirrored_dir_y[0] = 2;
//                mirrored_dir_y[1] = 1;
//                mirrored_dir_y[2] = 0;
//                mirrored_dir_y[3] = 3;

//                break;
//            }

//            // FHP model.
//            case 6:
//            {
//                // Define the memory offsets in the different directions for
//                // cells in rows with even and odd indices.
//                //
//                // The cell is located in a row with even index value.
//                if (pos_y % 2 == 0) {

//                    offset_to_neighbor[0] = 1;
//                    offset_to_neighbor[1] = n_x;
//                    offset_to_neighbor[2] = n_x - 1;
//                    offset_to_neighbor[3] = -1;
//                    offset_to_neighbor[4] = -n_x - 1;
//                    offset_to_neighbor[5] = -n_x;

//                    offset_to_eastern_boundary[0] = 0;
//                    offset_to_eastern_boundary[1] = 0;
//                    offset_to_eastern_boundary[2] = n_x;
//                    offset_to_eastern_boundary[3] = n_x;
//                    offset_to_eastern_boundary[4] = n_x;
//                    offset_to_eastern_boundary[5] = 0;

//                    offset_to_northern_boundary[0] = 0;
//                    offset_to_northern_boundary[1] = 0;
//                    offset_to_northern_boundary[2] = 0;
//                    offset_to_northern_boundary[3] = 0;
//                    offset_to_northern_boundary[4] = n_x * n_y;
//                    offset_to_northern_boundary[5] = n_x * n_y;

//                    offset_to_western_boundary[0] = -n_x;
//                    offset_to_western_boundary[1] = 0;
//                    offset_to_western_boundary[2] = 0;
//                    offset_to_western_boundary[3] = 0;
//                    offset_to_western_boundary[4] = 0;
//                    offset_to_western_boundary[5] = 0;

//                    offset_to_southern_boundary[0] = 0;
//                    offset_to_southern_boundary[1] = -n_x * n_y;
//                    offset_to_southern_boundary[2] = -n_x * n_y + 1;
//                    offset_to_southern_boundary[3] = 0;
//                    offset_to_southern_boundary[4] = 0;
//                    offset_to_southern_boundary[5] = 0;

//                // The cell is located in a row with odd index value.
//                } else if (pos_y % 2 != 0) {

//                    offset_to_neighbor[0] = 1;
//                    offset_to_neighbor[1] = n_x + 1;
//                    offset_to_neighbor[2] = n_x;
//                    offset_to_neighbor[3] = -1;
//                    offset_to_neighbor[4] = -n_x;
//                    offset_to_neighbor[5] = -n_x + 1;

//                    offset_to_eastern_boundary[0] = 0;
//                    offset_to_eastern_boundary[1] = 0;
//                    offset_to_eastern_boundary[2] = 0;
//                    offset_to_eastern_boundary[3] = n_x;
//                    offset_to_eastern_boundary[4] = 0;
//                    offset_to_eastern_boundary[5] = 0;

//                    offset_to_northern_boundary[0] = 0;
//                    offset_to_northern_boundary[1] = 0;
//                    offset_to_northern_boundary[2] = 0;
//                    offset_to_northern_boundary[3] = 0;
//                    offset_to_northern_boundary[4] = n_x * n_y;
//                    offset_to_northern_boundary[5] = n_x * n_y;

//                    offset_to_western_boundary[0] = -n_x;
//                    offset_to_western_boundary[1] = -n_x;
//                    offset_to_western_boundary[2] = 0;
//                    offset_to_western_boundary[3] = 0;
//                    offset_to_western_boundary[4] = 0;
//                    offset_to_western_boundary[5] = -n_x;

//                    offset_to_southern_boundary[0] = 0;
//                    offset_to_southern_boundary[1] = -n_x * n_y;
//                    offset_to_southern_boundary[2] = -n_x * n_y;
//                    offset_to_southern_boundary[3] = 0;
//                    offset_to_southern_boundary[4] = 0;
//                    offset_to_southern_boundary[5] = 0;
//                }

//                inverse_dir[0] = 3;
//                inverse_dir[1] = 4;
//                inverse_dir[2] = 5;
//                inverse_dir[3] = 0;
//                inverse_dir[4] = 1;
//                inverse_dir[5] = 2;

//                mirrored_dir_x[0] = 0;
//                mirrored_dir_x[1] = 5;
//                mirrored_dir_x[2] = 4;
//                mirrored_dir_x[3] = 3;
//                mirrored_dir_x[4] = 2;
//                mirrored_dir_x[5] = 1;

//                mirrored_dir_y[0] = 3;
//                mirrored_dir_y[1] = 2;
//                mirrored_dir_y[2] = 1;
//                mirrored_dir_y[3] = 0;
//                mirrored_dir_y[4] = 5;
//                mirrored_dir_y[5] = 4;

//                break;
//            }

//#ifdef DEBUG
//            default:
//            {
//                printf("ERROR in collide_and_propagate_kernel(): Invalid number of directions %d!\n", n_dir);
//                abort();
//                break;
//            }
//#endif

//        }

//        // Get index of the cell to work on.
//        int cell = n_x * blockIdx.y + pos_x;

//        // Get the type of the cell, i.e. fluid or solid.
//        // This has to be taken into account during the collision step, where
//        // cells behave different according to their type.
//        char cell_type = cell_type_gpu[cell];

//        // Check weather the cell is located on boundaries.
//        bool on_eastern_boundary  = ((blockIdx.x == (gridDim.x - 1)) && (pos_x == (n_x - 1)));
//        bool on_northern_boundary = (blockIdx.y == (gridDim.y - 1));
//        bool on_western_boundary  = ((threadIdx.x == 0) && (blockIdx.x == 0));
//        bool on_southern_boundary = (blockIdx.y == 0);

//        // Define an array for the global indices of the nodes in the cell.
//        int node_idx[n_dir];

//        // Define an array for the states of the nodes in the cell.
//        char node_state[n_dir];

//        // Execute collision step.
//        //
//        // The thread working on the cell has to know about the states of the
//        // nodes within the cell, therefore looping over all directions and
//        // look it up.
//#pragma unroll
//        for (int dir = 0; dir < n_dir; ++dir) {

//            node_idx[dir] = cell + dir * n_cells;
//            node_state[dir] = node_state_gpu[node_idx[dir]];
//        }

//        // TODO: Create a random boolean value for the collision step.
//        // bool rand_bool = cu_random_bool(seed, cell);
//        bool rand_bool =       ((pos_x % 2) == (pos_y % 2))
//                         - 1 * ((pos_x % 2) == (pos_y % 2)) * (step % 2)
//                         + 1 * ((pos_x % 2) != (pos_y % 2)) * (step % 2);

//        // Create a temporary array to copy the node states into.
//        char node_state_tmp[n_dir];

//        // Copy the actual states of the nodes to the temporary array.
//#pragma unroll
//        for (int dir = 0; dir < n_dir; ++dir) {

//            node_state_tmp[dir] = node_state[dir];
//        }

//        switch (cell_type) {

//            // The cell working on is a fluid cell ("normal" collision).
//            case 0:
//            {
//                // Using the the HPP model.
//                if (n_dir == 4) {

////                    // Collision case 1.
////                    if ((node_state[0] == 0) &&
////                        (node_state[1] == 1) &&
////                        (node_state[2] == 0) &&
////                        (node_state[3] == 1)) {
////
////                        node_state_tmp[0] = 1;
////                        node_state_tmp[1] = 0;
////                        node_state_tmp[2] = 1;
////                        node_state_tmp[3] = 0;
////
////                        break;
////                    }
////
////                    // Collision case 2.
////                    if ((node_state[0] == 1) &&
////                        (node_state[1] == 0) &&
////                        (node_state[2] == 1) &&
////                        (node_state[3] == 0)) {
////
////                        node_state_tmp[0] = 0;
////                        node_state_tmp[1] = 1;
////                        node_state_tmp[2] = 0;
////                        node_state_tmp[3] = 1;
////
////                        break;
////                    }

//                    node_state_tmp[0] = node_state[0]
//                            - (node_state[0] * node_state[2] * (1 - node_state[1]) * (1 - node_state[3]))
//                            + (node_state[1] * node_state[3] * (1 - node_state[0]) * (1 - node_state[2]));

//                    node_state_tmp[1] = node_state[1]
//                            - (node_state[1] * node_state[3] * (1 - node_state[0]) * (1 - node_state[2]))
//                            + (node_state[0] * node_state[2] * (1 - node_state[1]) * (1 - node_state[3]));

//                    node_state_tmp[2] = node_state[2]
//                            - (node_state[0] * node_state[2] * (1 - node_state[1]) * (1 - node_state[3]))
//                            + (node_state[1] * node_state[3] * (1 - node_state[0]) * (1 - node_state[2]));

//                    node_state_tmp[3] = node_state[3]
//                            - (node_state[1] * node_state[3] * (1 - node_state[0]) * (1 - node_state[2]))
//                            + (node_state[0] * node_state[2] * (1 - node_state[1]) * (1 - node_state[3]));

//                // Collision cases of the FHP model.
//                } else if (n_dir == 6) {

//                    // Collision case a1.
//                    if ((node_state[0] == 1) &&
//                        (node_state[1] == 0) &&
//                        (node_state[2] == 0) &&
//                        (node_state[3] == 1) &&
//                        (node_state[4] == 0) &&
//                        (node_state[5] == 0)) {

//                        node_state_tmp[0] = 0;
//                        node_state_tmp[1] = rand_bool;
//                        node_state_tmp[2] = 1 - node_state_tmp[1];
//                        node_state_tmp[3] = 0;
//                        node_state_tmp[4] = node_state_tmp[1];
//                        node_state_tmp[5] = node_state_tmp[2];

//                        break;
//                    }

//                    // Collision case a2.
//                    if ((node_state[0] == 0) &&
//                        (node_state[1] == 1) &&
//                        (node_state[2] == 0) &&
//                        (node_state[3] == 0) &&
//                        (node_state[4] == 1) &&
//                        (node_state[5] == 0)) {

//                        node_state_tmp[0] = rand_bool;
//                        node_state_tmp[1] = 0;
//                        node_state_tmp[2] = 1 - node_state_tmp[0];
//                        node_state_tmp[3] = node_state_tmp[0];
//                        node_state_tmp[4] = 0;
//                        node_state_tmp[5] = node_state_tmp[2];

//                        break;
//                    }

//                    // Collision case a3.
//                    if ((node_state[0] == 0) &&
//                        (node_state[1] == 0) &&
//                        (node_state[2] == 1) &&
//                        (node_state[3] == 0) &&
//                        (node_state[4] == 0) &&
//                        (node_state[5] == 1)) {

//                        node_state_tmp[0] = rand_bool;
//                        node_state_tmp[1] = 1 - node_state_tmp[0];
//                        node_state_tmp[2] = 0;
//                        node_state_tmp[3] = node_state_tmp[0];
//                        node_state_tmp[4] = node_state_tmp[1];
//                        node_state_tmp[5] = 0;

//                        break;
//                    }

//                    // Collision case b1.
//                    if ((node_state[0] == 0) &&
//                        (node_state[1] == 1) &&
//                        (node_state[2] == 0) &&
//                        (node_state[3] == 1) &&
//                        (node_state[4] == 0) &&
//                        (node_state[5] == 1)) {

//                        node_state_tmp[0] = 1;
//                        node_state_tmp[1] = 0;
//                        node_state_tmp[2] = 1;
//                        node_state_tmp[3] = 0;
//                        node_state_tmp[4] = 1;
//                        node_state_tmp[5] = 0;

//                        break;
//                    }

//                    // Collision case b2.
//                    if ((node_state[0] == 1) &&
//                        (node_state[1] == 0) &&
//                        (node_state[2] == 1) &&
//                        (node_state[3] == 0) &&
//                        (node_state[4] == 1) &&
//                        (node_state[5] == 0)) {

//                        node_state_tmp[0] = 0;
//                        node_state_tmp[1] = 1;
//                        node_state_tmp[2] = 0;
//                        node_state_tmp[3] = 1;
//                        node_state_tmp[4] = 0;
//                        node_state_tmp[5] = 1;

//                        break;
//                    }

////                    node_state_tmp[0] = node_state[0]
////                            - (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5]))
////                            + (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4])) * (1 - rand_bool)
////                            - (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]))
////                            + (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]));
////
////                    node_state_tmp[1] = node_state[1]
////                            - (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5]))
////                            + (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4])) * (1 - rand_bool)
////                            - (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]))
////                            + (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]));
////
////                    node_state_tmp[2] = node_state[2]
////                            - (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4]))
////                            + (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5])) * (1 - rand_bool)
////                            - (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]))
////                            + (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]));
////
////                    node_state_tmp[3] = node_state[3]
////                            - (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5]))
////                            + (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4])) * (1 - rand_bool)
////                            - (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]))
////                            + (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]));
////
////                    node_state_tmp[4] = node_state[4]
////                            - (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5]))
////                            + (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4])) * (1 - rand_bool)
////                            - (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]))
////                            + (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]));
////
////                    node_state_tmp[5] = node_state[5]
////                            - (node_state[2] * node_state[5] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[1]) * (1 - node_state[4]))
////                            + (node_state[0] * node_state[3] * (1 - node_state[1]) * (1 - node_state[4]) * (1 - node_state[2]) * (1 - node_state[5])) * rand_bool
////                            + (node_state[1] * node_state[4] * (1 - node_state[0]) * (1 - node_state[3]) * (1 - node_state[2]) * (1 - node_state[5])) * (1 - rand_bool)
////                            - (node_state[1] * node_state[3] * node_state[5] * (1 - node_state[0]) * (1 - node_state[2]) * (1 - node_state[4]))
////                            + (node_state[0] * node_state[2] * node_state[4] * (1 - node_state[1]) * (1 - node_state[3]) * (1 - node_state[5]));
//                }

//#ifdef DEBUG
//                else {

//                    printf("ERROR in collide_and_propagate_kernel(): "
//                           "Invalid number of directions %d.\n", n_dir);
//                }
//#endif

//                break;
//            }

//            // The cell working on is a solid cell of bounce back type.
//            case 1:
//            {
//                // Loop over all directions.
//                // #pragma unroll
//                for (int dir = 0; dir < n_dir; ++dir) {

//                    // Exchange the states of the nodes with the the states of
//                    // the inverse directions.
//                    node_state_tmp[dir] = node_state[inverse_dir[dir]];
//                }

//                break;
//            }

//            // TODO: The cell working on is a solid cell of bounce forward type.
//            case 2:
//            {
//                // Loop over all directions.
//#pragma unroll
//                for (int dir = 0; dir < n_dir; ++dir) {

//                    if (on_northern_boundary || on_southern_boundary) {

//                        // Exchange the states of the nodes with the the states of
//                        // the mirrored directions along the x axis.
//                        node_state_tmp[dir] = node_state[mirrored_dir_x[dir]];
//                    }

//                    if (on_eastern_boundary || on_western_boundary) {

//                        // Exchange the states of the nodes with the the states of
//                        // the mirrored directions along the y axis.
//                        node_state_tmp[dir] = node_state[mirrored_dir_y[dir]];
//                    }
//                }

//                break;
//            }

//#ifdef DEBUG
//            // Invalid cell type.
//            default:
//            {
//                printf("ERROR in collide_and_propagate_kernel(): Invalid cell type %d.\n", cell_type);
//                break;
//            }
//#endif

//        }

//        // Execute propagation step.
//        //
//        // Loop over all directions.
//        // #pragma unroll
//        for (int dir = 0; dir < n_dir; dir++)
//        {
//            // Reset the memory offset.
//            int offset = 0;

//            // Construct the correct memory offset.
//            //
//            // Apply a default offset value.
//            offset += offset_to_neighbor[dir];

//            // Correct the offset in the current direction if the cell is
//            // located on boundaries.
//            if (on_eastern_boundary) {

//                offset += offset_to_western_boundary[dir];
//            }

//            if (on_northern_boundary) {

//                offset += offset_to_southern_boundary[dir];
//            }

//            if (on_western_boundary) {

//                offset += offset_to_eastern_boundary[dir];
//            }

//            if (on_southern_boundary) {

//                offset += offset_to_northern_boundary[dir];
//            }

//            // Push the states of the cell to its "neighbor" cells in the
//            // different directions.
//            node_state_tmp_gpu[node_idx[dir] + offset] = node_state_tmp[dir];
//        }
//    }
//}

//// Applies a body force in the specified direction (x or y) to the particles.
//template <int n_dir, char bf_dir>
//__global__ void apply_body_force_kernel(int   forcing,
//                                        int   n_x,
//                                        int   n_cells,
//                                        char* cell_type_gpu,
//                                        char* node_state_gpu,
//                                        char* node_state_tmp_gpu,
//                                        int   seed) {

//    // Each thread is looking for one particle to revert.

//    // Lattice vector components in the different directions.
//    Real lattice_vec_x[n_dir];
//    Real lattice_vec_y[n_dir];

//    // Mirrored direction indices for each lattice direction with respect
//    // to the x and y axis.
//    char mirrored_dir_x[n_dir];
//    char mirrored_dir_y[n_dir];

//    // Set the components of the lattice vectors for the different directions.
//    //
//    // Loop over all directions.
//    for (int dir = 0; dir < n_dir; ++dir) {

//        lattice_vec_x[dir] = cos(2.0 * M_PI / ((Real) n_dir) * ((Real) dir));
//        lattice_vec_y[dir] = sin(2.0 * M_PI / ((Real) n_dir) * ((Real) dir));
//    }

//    // Set the model based values according to the number of lattice directions.
//    switch (n_dir) {

//        // HPP model.
//        case 4:
//        {
//            mirrored_dir_x[0] = 0;
//            mirrored_dir_x[1] = 3;
//            mirrored_dir_x[2] = 2;
//            mirrored_dir_x[3] = 1;

//            mirrored_dir_y[0] = 2;
//            mirrored_dir_y[1] = 1;
//            mirrored_dir_y[2] = 0;
//            mirrored_dir_y[3] = 3;

//            break;
//        }

//        // FHP model.
//        case 6:
//        {
//            mirrored_dir_x[0] = 0;
//            mirrored_dir_x[1] = 5;
//            mirrored_dir_x[2] = 4;
//            mirrored_dir_x[3] = 3;
//            mirrored_dir_x[4] = 2;
//            mirrored_dir_x[5] = 1;

//            mirrored_dir_y[0] = 3;
//            mirrored_dir_y[1] = 2;
//            mirrored_dir_y[2] = 1;
//            mirrored_dir_y[3] = 0;
//            mirrored_dir_y[4] = 5;
//            mirrored_dir_y[5] = 4;

//            break;
//        }

//#ifdef DEBUG
//        default:
//        {
//            printf("ERROR in Lattice(): Invalid number of directions %d!\n", n_dir);
//            abort();
//            break;
//        }
//#endif

//    }

//    // Set a maximum number of iterations to find particles which can be reverted.
//    const int it_max = 2 * n_cells;

//    // Set the number of iterations to zero.
//    int it = 0;

//    // Number of particles which have been reverted.
//    int reverted_particles = 0;

//    // Loop over all cells.
//    do
//    {
//        // TODO: Get the index of a random cell.
//        int cell = (int)truncf(cu_random(seed, threadIdx.x) * n_cells);
//        printf("cell = %d", cell);
//        it++;

//        // Get the type of the cell, i.e. fluid or solid.
//        // Note that body forces are applied to fluid cells only.
//        char cell_type = cell_type_gpu[cell];

//        // Check weather the cell working on is a fluid cell.
//        if (cell_type == 0) {

//            // Define an array for the global indices of the nodes in the cell.
//            int node_idx[n_dir];

//            // Define an array for the states of the nodes in the cell.
//            char node_state[n_dir];

//            // The thread working on the cell has to know about the states of the
//            // nodes within the cell, therefore looping over all directions and
//            // look it up.
//        #pragma unroll
//            for (int dir = 0; dir < n_dir; ++dir) {

//                node_idx[dir] = cell + dir * n_cells;
//                node_state[dir] = node_state_gpu[node_idx[dir]];
//            }

//            // Create a temporary array to copy the node states into.
//            char node_state_tmp[n_dir];

//            // Copy the current states of the nodes to the temporary array.
//        #pragma unroll
//            for (int dir = 0; dir < n_dir; ++dir) {

//                node_state_tmp[dir] = node_state[dir];
//            }

//            if (n_dir == 4) {

//                if (bf_dir == 'x' && (node_state[0] == 0) && (node_state[2] == 1)) {

//                    node_state_tmp[0] = 1;
//                    node_state_tmp[2] = 0;

//                    reverted_particles++;

//                } else if (bf_dir == 'y' && (node_state[1] == 1) && (node_state[3] == 0)) {

//                    node_state_tmp[1] = 0;
//                    node_state_tmp[3] = 1;

//                    reverted_particles++;
//                }
//            }

//            else if (n_dir == 6) {

//                if (bf_dir == 'x' && (node_state[0] == 0) && (node_state[3] == 1)) {

//                    node_state_tmp[0] = 1;
//                    node_state_tmp[3] = 0;

//                    reverted_particles++;

//                } else if (bf_dir == 'y') {

//                    if ((node_state[1] == 1) && (node_state[5] == 0)) {

//                        node_state_tmp[1] = 0;
//                        node_state_tmp[5] = 1;

//                        reverted_particles++;
//                    }

//                    if ((node_state[2] == 1) && (node_state[4] == 0)) {

//                        node_state_tmp[2] = 0;
//                        node_state_tmp[4] = 1;

//                        reverted_particles++;
//                    }
//                }
//            }

//    //            // Loop over all directions.
//    //#pragma unroll
//    //            for (int dir = 0; dir < n_dir; ++dir) {
//    //
//    //                // Body force acting in x direction.
//    //                if (bf_dir == 'x') {
//    //
//    //					// TODO: Exchange the states of the nodes with the the states of
//    //					//       the mirrored directions along the y axis if feasible.
//    //					if ((fabs(lattice_vec_x[dir] - 1.0) < 1.0e-06) &&
//    //						(node_state[dir] < node_state[mirrored_dir_y[dir]])) {
//    //
//    //						node_state_tmp[dir                ] = node_state[mirrored_dir_y[dir]];
//    //						node_state_tmp[mirrored_dir_y[dir]] = node_state[dir                ];
//    //					}
//    //                }
//    //
//    //                // Body force acting in y direction.
//    //                else if (bf_dir == 'y') {
//    //
//    //					// TODO: Exchange the states of the nodes with the the states of
//    //					//       the mirrored directions along the x axis if feasible.
//    //					if ((lattice_vec_y[dir] < 1.0e-06) &&
//    //						(node_state[dir] < node_state[mirrored_dir_x[dir]])) {
//    //
//    //						node_state_tmp[dir                ] = node_state[mirrored_dir_x[dir]];
//    //						node_state_tmp[mirrored_dir_x[dir]] = node_state[dir                ];
//    //					}
//    //                }
//    //
//    //#ifdef DEBUG
//    //                // Invalid body force direction.
//    //                else {
//    //
//    //                    printf("ERROR in apply_body_force(): "
//    //                           "Invalid body force direction %c.\n", bf_dir);
//    //                }
//    //#endif
//    //            }

//            // Write the new node states back to the data array.
//            //
//            // Loop over all directions.
//        #pragma unroll
//            for(int dir = 0; dir < n_dir; dir++)
//            {
//                node_state_gpu[node_idx[dir]] = node_state_tmp[dir];
//            }

//        } /* IF cell_type */

//    } while ((reverted_particles < 1) && (it < it_max));
//}

//// Computes the mean velocity of the lattice.
//__global__ void get_mean_velocity_kernel(int *g_idata, int *g_odata) {

//    extern __shared__ int sdata[];

//    // Each thread loads one element from global to shared memory.
//    unsigned int tid = threadIdx.x;
//    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//    sdata[tid] = g_idata[i];
//    __syncthreads();

//    // Do reduction in shared memory.
//    for (unsigned int s = 1; s < blockDim.x; s *= 2) {

//        if (tid % (2*s) == 0) {

//            sdata[tid] += sdata[tid + s];
//        }
//        __syncthreads();
//    }

//    // Write result for this block to global memory.
//    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
//}

//// Computes cell quantities of interest as a post-processing procedure.
//template <int n_dir>
//__global__ void cell_post_process_kernel(const int     n_x,
//                                         const int     n_cells,
//                                         const char*   node_state_gpu,
//                                         Real*         cell_density_gpu,
//                                         Real*         cell_momentum_gpu) {

//    // Each thread is working on one cell of the lattice.

//    // Thread is active per default.
//    bool active = true;

//    // Calculate the position of the cell in x direction.
//    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;

//    // Check weather the thread is working on a valid cell.
//    if (pos_x >= n_x) {

//        active = false;
//    }

//    // Start calculation only for activated threads working on valid cells.
//    if (active) {

//        // Lattice vector components in the different directions.
//        Real lattice_vec_x[n_dir];
//        Real lattice_vec_y[n_dir];

//        // Set the components of the lattice vectors for the different directions.
//        //
//        // Loop over all directions.
//        for (int dir = 0; dir < n_dir; ++dir) {

//            lattice_vec_x[dir] = cos(2.0 * M_PI / ((Real) n_dir) * ((Real) dir));
//            lattice_vec_y[dir] = sin(2.0 * M_PI / ((Real) n_dir) * ((Real) dir));
//        }

//        // Get index of the cell to work on.
//        int cell = n_x * blockIdx.y + pos_x;

//        // Define an array for the global indices of the nodes in a cell.
//        int node_idx[n_dir];

//        // Define an array for the states of the nodes in a cell.
//        char node_state[n_dir];

//        // The thread working on the cell has to know about the states of the
//        // nodes within the cell, therefore looping over all directions and
//        // look it up.
//#pragma unroll
//        for (int dir = 0; dir < n_dir; ++dir) {

//            node_idx[dir] = cell + dir * n_cells;
//            node_state[dir] = node_state_gpu[node_idx[dir]];
//        }

//        // Initialize the cell quantities to be computed.
//        int  cell_density    = 0;
//        Real cell_momentum_x = 0.0;
//        Real cell_momentum_y = 0.0;

//        // Loop over all nodes in the cell.
//#pragma unroll
//        for (int dir = 0; dir < n_dir; ++dir) {

//            // Sum up the node states.
//            cell_density += node_state[dir];

//            // Sum up the node states multiplied by the lattice vector component
//            // for the current direction.
//            cell_momentum_x += node_state[dir] * lattice_vec_x[dir];
//            cell_momentum_y += node_state[dir] * lattice_vec_y[dir];
//        }

//        // Write the computed cell quantities to the related data arrays.
//        cell_density_gpu [cell          ] = (Real) cell_density;
//        cell_momentum_gpu[cell          ] =        cell_momentum_x;
//        cell_momentum_gpu[cell + n_cells] =        cell_momentum_y;
//    }
//}

//// Computes coarse grained quantities of interest as a post-processing procedure.
//__global__ void mean_post_process_kernel(const int   n_x,
//                                         const int   n_cells,
//                                         const int   coarse_graining_radius,
//                                         const Real* cell_density_gpu,
//                                         const Real* cell_momentum_gpu,
//                                               Real* mean_density_gpu,
//                                               Real* mean_momentum_gpu) {

//    // Each thread is working on one cell of the lattice.

//    // Thread is active per default.
//    bool active = true;

//    // Calculate the position of the cell in x direction.
//    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;

//    // Check weather the thread is working on a valid cell.
//    if (pos_x >= n_x) {

//        active = false;
//    }

//    // Start calculation only for activated threads working on valid cells.
//    if (active) {

//        // Get index of the cell to work on.
//        int cell = n_x * blockIdx.y + pos_x;

//        // Initialize the coarse grained quantities to be computed.
//        Real mean_density    = 0.0;
//        Real mean_momentum_x = 0.0;
//        Real mean_momentum_y = 0.0;

//        // Initialize the number of actual existing coarse graining neighbor cells.
//        int n_exist_neighbors = 0;

//        // The thread working on the cell has to know the cell quantities of the
//        // coarse graining neighbor cells, therefore looping over all neighbor
//        // cells and look it up.
//#pragma unroll
//        for (int y = -coarse_graining_radius; y <= coarse_graining_radius; ++y) {

//            for (int x = -coarse_graining_radius; x <= coarse_graining_radius; ++x) {

//                // Get the index of the coarse graining neighbor cell.
//                int neighbor_idx = cell + y * n_x + x;

//                // Get the position of the coarse graining neighbor cell in x direction.
//                int pos_x_neighbor = neighbor_idx % n_x;

//                // Check weather the coarse graining neighbor cell is valid.
//                if ((neighbor_idx >= 0) &&
//                    (neighbor_idx < n_cells) &&
//                    (abs(pos_x_neighbor - pos_x) <= coarse_graining_radius)) {

//                    // Increase the number of existing coarse graining neighbor cells.
//                    n_exist_neighbors++;

//                    mean_density    += cell_density_gpu [neighbor_idx          ];
//                    mean_momentum_x += cell_momentum_gpu[neighbor_idx          ];
//                    mean_momentum_y += cell_momentum_gpu[neighbor_idx + n_cells];
//                }
//            }
//        }

//        // Write the computed coarse grained quantities to the related data arrays.
//        mean_density_gpu [cell          ] = mean_density    / ((Real) n_exist_neighbors);
//        mean_momentum_gpu[cell          ] = mean_momentum_x / ((Real) n_exist_neighbors);
//        mean_momentum_gpu[cell + n_cells] = mean_momentum_y / ((Real) n_exist_neighbors);
//    }
//}

//// Creates a CUDA parallelized lattice gas cellular automaton object
//// of the specified properties.
//CUDA_Lattice::CUDA_Lattice(const string test_case,
//                           const Real Re, const Real Ma_s,
//                           const int n_dir,
//                           const int coarse_graining_radius,
//                           const int device = 0)

//                           : Lattice(test_case, Re, Ma_s, n_dir, coarse_graining_radius) {

//    // Set the device to use for the simulation.
//    int n_devices;
//    cudaGetDeviceCount(&n_devices);
//    assert((device < n_devices) && (device >= 0));
//    this->device = device;
//    cudaSetDevice(device);

//    // Allocate the memory for the arrays on the host (CPU) and device (GPU).
//    allocate_memory();
//}

//// Deletes the CUDA parallelized lattice gas cellular automaton object.
//CUDA_Lattice::~CUDA_Lattice() {

//    free_memory();
//}

//// Sets (proper) grid and block sizes for the GPU computation.
//void CUDA_Lattice::set_grid_and_block_size(int max_block_size = 256) {

//    grid_size_x = 1;
//    grid_size_y = n_y;
//    grid_size_z = 1;

//    block_size_x = n_x;
//    block_size_y = 1;
//    block_size_z = 1;

//    if (n_x > max_block_size) {

//        grid_size_x  = (int) (ceil((Real)n_x / (Real)max_block_size) + 0.5);
//        block_size_x = max_block_size;
//        if (n_x % max_block_size != 0) {
//            printf("WARNING in Lattice::set_grid_and_block_size(): "
//                   "There are inactive threads in some blocks!\n");
//        }
//    }

//    cudaDeviceProp prop;
//    cudaGetDeviceProperties(&prop, device);

//    if ( grid_size_x                               <=   prop.maxGridSize[0] &&
//         grid_size_y                               <=   prop.maxGridSize[1] &&
//         grid_size_z                               <=   prop.maxGridSize[2] &&
//        block_size_x                               <= prop.maxThreadsDim[0] &&
//        block_size_y                               <= prop.maxThreadsDim[1] &&
//        block_size_z                               <= prop.maxThreadsDim[2] &&
//        block_size_x * block_size_y * block_size_z <= prop.maxThreadsPerBlock) {

//        printf("Kernel configuration parameters: %d x %d x %d Blocks \n",   grid_size_x , grid_size_y , grid_size_z );
//        printf("                                 %d x %d x %d Threads\n\n", block_size_x, block_size_y, block_size_z);

//    } else {

//        printf("ERROR in Lattice::set_grid_and_block_size():"
//               "Invalid grid and/or block dimensions. "
//               "Please check device properties.");
//        abort();
//    }
//}

//// Copies all data arrays from the device (GPU) back to the host (CPU).
//void CUDA_Lattice::copy_data_from_device() {

//    cu_verify(cudaMemcpy(node_state_cpu,    node_state_gpu,          n_nodes * sizeof(char), cudaMemcpyDeviceToHost));
//    cu_verify(cudaMemcpy(cell_density_cpu,  cell_density_gpu,        n_cells * sizeof(Real), cudaMemcpyDeviceToHost));
//    cu_verify(cudaMemcpy(mean_density_cpu,  mean_density_gpu,        n_cells * sizeof(Real), cudaMemcpyDeviceToHost));
//    cu_verify(cudaMemcpy(cell_momentum_cpu, cell_momentum_gpu, dim * n_cells * sizeof(Real), cudaMemcpyDeviceToHost));
//    cu_verify(cudaMemcpy(mean_momentum_cpu, mean_momentum_gpu, dim * n_cells * sizeof(Real), cudaMemcpyDeviceToHost));
//}

//// Copies all data arrays from the host (CPU) to the device (GPU).
//void CUDA_Lattice::copy_data_to_device() {

//    cu_verify(cudaMemcpy(cell_type_gpu,  cell_type_cpu,  n_cells * sizeof(char), cudaMemcpyHostToDevice));
//    cu_verify(cudaMemcpy(node_state_gpu, node_state_cpu, n_nodes * sizeof(char), cudaMemcpyHostToDevice));
//}

//// Allocates the memory for the arrays on the host (CPU) and device (GPU).
//void CUDA_Lattice::allocate_memory() {

//    // Allocate host memory.
//    cu_verify(cudaMallocHost((void **) &node_state_cpu,          n_nodes * sizeof(char)));
//    cu_verify(cudaMallocHost((void **) &cell_type_cpu,           n_cells * sizeof(char)));
//    cu_verify(cudaMallocHost((void **) &cell_density_cpu,        n_cells * sizeof(Real)));
//    cu_verify(cudaMallocHost((void **) &mean_density_cpu,        n_cells * sizeof(Real)));
//    cu_verify(cudaMallocHost((void **) &cell_momentum_cpu, dim * n_cells * sizeof(Real)));
//    cu_verify(cudaMallocHost((void **) &mean_momentum_cpu, dim * n_cells * sizeof(Real)));

//    // Allocate device memory.
//    cu_verify(cudaMalloc((void **) &node_state_gpu,          n_nodes * sizeof(char)));
//    cu_verify(cudaMalloc((void **) &node_state_tmp_gpu,      n_nodes * sizeof(char)));
//    cu_verify(cudaMalloc((void **) &cell_type_gpu,           n_cells * sizeof(char)));
//    cu_verify(cudaMalloc((void **) &cell_density_gpu,        n_cells * sizeof(Real)));
//    cu_verify(cudaMalloc((void **) &mean_density_gpu,        n_cells * sizeof(Real)));
//    cu_verify(cudaMalloc((void **) &cell_momentum_gpu, dim * n_cells * sizeof(Real)));
//    cu_verify(cudaMalloc((void **) &mean_momentum_gpu, dim * n_cells * sizeof(Real)));
//}

//// Frees the memory for the arrays on the host (CPU) and device (GPU).
//void CUDA_Lattice::free_memory() {

//    // Free GPU memory.
//    cu_verify(cudaFree(node_state_gpu));
//    cu_verify(cudaFree(node_state_tmp_gpu));
//    cu_verify(cudaFree(cell_type_gpu));
//    cu_verify(cudaFree(cell_density_gpu));
//    cu_verify(cudaFree(mean_density_gpu));
//    cu_verify(cudaFree(cell_momentum_gpu));
//    cu_verify(cudaFree(mean_momentum_gpu));

//    // Free CPU memory.
//    cu_verify(cudaFreeHost(node_state_cpu));
//    cu_verify(cudaFreeHost(cell_type_cpu));
//    cu_verify(cudaFreeHost(cell_density_cpu));
//    cu_verify(cudaFreeHost(mean_density_cpu));
//    cu_verify(cudaFreeHost(cell_momentum_cpu));
//    cu_verify(cudaFreeHost(mean_momentum_cpu));
//}

//// Calls the CUDA kernel which performs the collision and propagation step
//// on the lattice gas automaton.
//void CUDA_Lattice::collide_and_propagate(unsigned int step) {

//    // Set the grid and block size.
//    dim3 grid_size (grid_size_x,  grid_size_y,  grid_size_z);
//    dim3 block_size(block_size_x, block_size_y, block_size_z);

//    // TODO: Set the seed for the random number generation on the device.
//    int seed = time(NULL);

//    // Call CUDA kernel.
//    switch (n_dir) {

//        // HPP model.
//        case 4:
//        {
//            cu_verify_void((collide_and_propagate_kernel<4>
//                    <<<grid_size, block_size>>>(n_x,
//                                                n_y,
//                                                n_cells,
//                                                cell_type_gpu,
//                                                node_state_gpu,
//                                                node_state_tmp_gpu,
//                                                step)));
//            break;
//        }

//        // FHP model.
//        case 6:
//        {
//            cu_verify_void((collide_and_propagate_kernel<6>
//                    <<<grid_size, block_size>>>(n_x,
//                                                n_y,
//                                                n_cells,
//                                                cell_type_gpu,
//                                                node_state_gpu,
//                                                node_state_tmp_gpu,
//                                                step)));
//            break;
//        }
//        default:
//        {
//            printf("ERROR in collide_and_propagate(): "
//                   "Invalid number of directions %d.\n", n_dir);
//            abort();

//            break;
//        }
//    }

//    // Wait for all threads to finish.
//    cudaDeviceSynchronize();

//    // Update the node states.
//    char* node_state_gpu_tmp = node_state_gpu;
//    node_state_gpu = node_state_tmp_gpu;
//    node_state_tmp_gpu = node_state_gpu_tmp;
//}

//// Calls the CUDA kernel which applies a body force in the specified
//// direction (x or y) to the particles.
//void CUDA_Lattice::apply_body_force(const int forcing) {

//    const int max_block_size = 256;

//    // Set the grid and block size.
//    int grid_size_x = forcing / max_block_size + 1;

//    int block_size_x = 1;
//    if (forcing > 0)   block_size_x = forcing;
//    if (forcing > 256) block_size_x = max_block_size;

//    dim3 grid_size (grid_size_x,  1, 1);
//    dim3 block_size(block_size_x, 1, 1);

//    // Get device properties.
//    cudaDeviceProp prop;
//    cudaGetDeviceProperties(&prop, device);

//    if ( grid_size.x                               <= prop.maxGridSize[0] &&
//         grid_size.y                               <= prop.maxGridSize[1] &&
//         grid_size.z                               <= prop.maxGridSize[2] &&
//        block_size.x                               <= prop.maxThreadsDim[0] &&
//        block_size.y                               <= prop.maxThreadsDim[1] &&
//        block_size.z                               <= prop.maxThreadsDim[2] &&
//        block_size.x * block_size.y * block_size.z <= prop.maxThreadsPerBlock) {

//        printf("Kernel configuration parameters: %d x %d x %d Blocks \n", grid_size.x , grid_size.y , grid_size.z );
//        printf("                                 %d x %d x %d Threads\n", block_size.x, block_size.y, block_size.z);

//    } else {

//        printf("ERROR in CUDA_Lattice::apply_body_force():"
//               "Invalid grid and/or block dimensions. "
//               "Please check device properties.");
//        abort();
//    }

//    // TODO: Set the seed for the random number generation on the device.
//    int seed = time(NULL);

//    // Call CUDA kernel.
//    switch (n_dir) {

//        // HPP model.
//        case 4:
//        {
//            switch (bf_dir) {

//                // Apply body force in x direction.
//                case 'x':
//                {
//                    cu_verify_void((apply_body_force_kernel<4, 'x'>
//                            <<<grid_size, block_size>>>(forcing,
//                                                        n_x,
//                                                        n_cells,
//                                                        cell_type_gpu,
//                                                        node_state_gpu,
//                                                        node_state_tmp_gpu,
//                                                        seed)));
//                    break;
//                }

//                // Apply body force in y direction.
//                case 'y':
//                {
//                    cu_verify_void((apply_body_force_kernel<4, 'y'>
//                            <<<grid_size, block_size>>>(forcing,
//                                                        n_x,
//                                                        n_cells,
//                                                        cell_type_gpu,
//                                                        node_state_gpu,
//                                                        node_state_tmp_gpu,
//                                                        seed)));
//                    break;
//                }

//                // Invalid body force direction.
//                default:
//                {
//                    printf("ERROR in apply_body_force(): "
//                           "Invalid body force direction %c.\n", bf_dir);
//                    abort();

//                    break;
//                }
//            }

//            break;
//        }

//        // FHP model.
//        case 6:
//        {
//            switch (bf_dir) {

//                // Apply body force in x direction.
//                case 'x':
//                {
//                    cu_verify_void((apply_body_force_kernel<6, 'x'>
//                            <<<grid_size, block_size>>>(forcing,
//                                                        n_x,
//                                                        n_cells,
//                                                        cell_type_gpu,
//                                                        node_state_gpu,
//                                                        node_state_tmp_gpu,
//                                                        seed)));
//                    break;
//                }

//                // Apply body force in y direction.
//                case 'y':
//                {
//                    cu_verify_void((apply_body_force_kernel<6, 'y'>
//                            <<<grid_size, block_size>>>(forcing,
//                                                        n_x,
//                                                        n_cells,
//                                                        cell_type_gpu,
//                                                        node_state_gpu,
//                                                        node_state_tmp_gpu,
//                                                        seed)));
//                    break;
//                }

//                // Invalid body force direction.
//                default:
//                {
//                    printf("ERROR in apply_body_force(): "
//                           "Invalid body force direction %c.\n", bf_dir);
//                    abort();

//                    break;
//                }
//            }

//            break;
//        }

//        // Invalid number of directions.
//        default:
//        {
//            printf("ERROR in apply_body_force(): "
//                   "Invalid number of directions %d.\n", n_dir);
//            abort();

//            break;
//        }
//    }


//    // Wait for all threads to finish.
//    cudaDeviceSynchronize();
//}

//// Call the CUDA kernel which computes quantities of interest as a
//// post-processing procedure.
//void CUDA_Lattice::post_process() {

//    // Set the grid and block size.
//    dim3 grid_size (grid_size_x,  grid_size_y,  grid_size_z);
//    dim3 block_size(block_size_x, block_size_y, block_size_z);

//    // Call CUDA kernel.
//    switch (n_dir) {

//        // HPP model.
//        case 4:
//        {
//            cu_verify_void((cell_post_process_kernel<4>
//                    <<<grid_size, block_size>>>(n_x,
//                                                n_cells,
//                                                node_state_gpu,
//                                                cell_density_gpu,
//                                                cell_momentum_gpu)));
//            break;
//        }

//        // FHP model.
//        case 6:
//        {
//            cu_verify_void((cell_post_process_kernel<6>
//                    <<<grid_size, block_size>>>(n_x,
//                                                n_cells,
//                                                node_state_gpu,
//                                                cell_density_gpu,
//                                                cell_momentum_gpu)));
//            break;
//        }

//        // Invalid number of directions.
//        default:
//        {
//            printf("ERROR in post_process(): "
//                   "Invalid number of directions %d.\n", n_dir);
//            abort();

//            break;
//        }
//    }

//    // Wait for all threads to finish.
//    cudaDeviceSynchronize();

//    cu_verify_void((mean_post_process_kernel
//            <<<grid_size, block_size>>>(n_x,
//                                        n_cells,
//                                        coarse_graining_radius,
//                                        cell_density_gpu,
//                                        cell_momentum_gpu,
//                                        mean_density_gpu,
//                                        mean_momentum_gpu)));

//    // Wait for all threads to finish.
//    cudaDeviceSynchronize();
//}

//// Sets (proper) parallelization parameters.
//void CUDA_Lattice::setup_parallel()
//{
//    // Sets (proper) grid and block size for the GPU computation.
//    set_grid_and_block_size(256);
//}

//// TODO: Computes the mean velocity of the lattice.
//std::vector<Real> CUDA_Lattice::get_mean_velocity()
//{
//    std::vector<Real> mean_velocity(dim, 0.0);

//    Real sum_x_vel = 0.0;
//    Real sum_y_vel = 0.0;

//    unsigned int counter = 0;

//    // Sum up all (fluid) cell x and y velocity components.
//#pragma omp parallel for reduction(+: sum_x_vel, sum_y_vel)
//    for (unsigned int n = 0; n < n_cells; ++n) {

//        if (cell_type_cpu[n] == 0) {

//            counter++;

//            Real cell_density = cell_density_cpu[n];

//            if (cell_density > 1.0e-06) {

//                sum_x_vel += cell_momentum_cpu[n          ] / cell_density;
//                sum_y_vel += cell_momentum_cpu[n + n_cells] / cell_density;
//            }

//#ifdef DEBUG

//            else if (fabs(cell_density) < 1.0e-06) {

//                // Do nothing.

//            } else if (cell_density < -1.0e-06) {

//                printf("ERROR in get_mean_velocity(): "
//                       "Negative cell density detected.");
//                abort();
//            }

//#endif

//        }
//    }

//    // Divide the summed up x and y components by the total number of fluid cells.
//    mean_velocity[0] = sum_x_vel / (Real) counter;
//    mean_velocity[1] = sum_y_vel / (Real) counter;

//    return mean_velocity;
//}

